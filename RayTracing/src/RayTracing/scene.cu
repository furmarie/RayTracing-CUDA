#include "hip/hip_runtime.h"
#include <chrono>
//#include <thread>
//
//#include "ctpl.h"
#include "scene.cuh"
#include "ray.hpp"
//#include "materials/simplematerial.hpp"
//#include "textures/checker.hpp"
#include <cassert>
#include <iostream>
#include <iomanip>


#define checkCudaErrors1(val) check_cuda1( (val), #val, __FILE__, __LINE__ )
void check_cuda1(hipError_t result, char const* const func, const char* const file, int const line) {
	if(result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		auto errName = hipGetErrorName(result);
		std::cerr << errName << '\n';
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}


//__device__
//vec3 pixel_func(const int x, const int y, const int xSize, const int ySize, camera** cam) {
//	float xFact = 1.0 / (static_cast<float>(xSize) / 2.0);
//	float yFact = 1.0 / (static_cast<float>(ySize) / 2.0);
//	float minDist = 1e6;
//	float maxDist = 0.0;
//
//	float normX = (static_cast<float>(x) * xFact) - 1.0;
//	float normY = (static_cast<float>(y) * yFact) - 1.0;
//
//	// Check each pixel in image and send ray out to that
//	// and check if there is an intersection
//	fRT::ray cameraRay;
//
//	// Generate the ray for this pixel.
//	(*cam)->generateRay(normX, normY, cameraRay);
//
//	// Hold records of closest hit object
//	//hitRecord record;
//	//std::shared_ptr<objectBase> currObj = nullptr;
//
//	int currDepth = 0;
//
//	vec3 colour;
//
//	// Attenuates next colour after reflection
//	float attenuation = 1.0;
//
//	vec3 cameraScreenCentre = (*cam)->getScreenCentre();
//	/*
//	while(currDepth++ < maxReflectionDepth and attenuation > 0.0) {
//		bool intersectionFound = materialBase::castRay(
//			cameraRay,
//			m_world,
//			currObj,
//			record
//		);
//
//
//
//		// Compute illumination for closest object if intersectionFound
//		if(intersectionFound) {
//			//exit(0);
//			currObj = record.obj;
//			// Check if the closest object has a material
//			if(currObj->m_hasMaterial) {
//				// Get colour components without reflective component of this material
//				float currObjReflectivity = 0.0;
//				vec3 colourComps = record.obj->m_pMaterial->getColour(
//					m_world,
//					m_lights,
//					record,
//					cameraRay,
//					currObjReflectivity
//				);
//
//				colour += attenuation * colourComps;
//				attenuation *= currObjReflectivity;
//
//				// Change cameraRay to reflected ray at this position
//				// Compute the reflection vector
//				vec3 d = cameraRay.m_AB;
//				vec3 reflectedVector = d - 2 * (vec3::dot(d, record.localNormal)) * record.localNormal;
//
//				// Construct the reflection ray
//				vec3 startPoint = record.intPoint;
//				ray reflectedRay(startPoint, startPoint + reflectedVector);
//				cameraRay = reflectedRay;
//			}
//			else {
//				// Use basic method to compute colour as object has no material
//				vec3 matColour = materialBase::computeDiffuseColour(m_world, m_lights, record, record.obj->m_baseColour);
//
//				colour += attenuation * matColour;
//
//				break; // NO more reflection
//
//				//outputImage.SetPixel(x, y, matColour[0], matColour[1], matColour[2]);
//			}
//		}
//		else {
//			// No intersection after this
//			//
//			// Get Sky Colour from direction of ray
//
//			// Direction vector is somewhere on projection screen
//			// The Z value ranges from centre - VertSize to centre + VertSize
//			float vertSize = m_camera.getVertSize();
//			auto z = cameraRay.m_AB[2];
//			float t = (z - (cameraScreenCentre[2] - vertSize)) / (2 * vertSize);
//
//			//float t = 0.5 * (-dir_unit_vec[2] + 1.0);
//			//if(t < 0.0 or t > 1.0) {
//			//	std::cerr << t << '\n';
//			//	exit(0);
//			//}
//
//			t = fmax(t, 0.0);
//			t = fmin(t, 1.0);
//
//			//std::cerr << t << std::endl;
//			//if(x == xSize - 2 and y < ySize - 10) {
//			//	exit(0);
//			//}
//
//			mn = fmin(mn, t);
//			mx = fmax(mx, t);
//
//			vec3 initial({ 1, 1, 1 });
//			vec3 final_colour({ 0.5, 0.7, 1.0 });
//			colour += attenuation * (t * initial + (1 - t) * final_colour);
//			break;
//		}
//	}
//	*/
//
//	// Get Sky Colour from direction of ray
//
//			// Direction vector is somewhere on projection screen
//			// The Z value ranges from centre - VertSize to centre + VertSize
//	float vertSize = (*cam)->getVertSize();
//	auto z = cameraRay.m_AB.z;
//	float t = (z - (cameraScreenCentre.z - vertSize)) / (2 * vertSize);
//
//	//float t = 0.5 * (-dir_unit_vec[2] + 1.0);
//	//if(t < 0.0 or t > 1.0) {
//	//	std::cerr << t << '\n';
//	//	exit(0);
//	//}
//	if(t < 0.0) {
//		t = 0.0;
//	}
//	if(t > 1.0) {
//		t = 1.0;
//	}
//
//
//	vec3 initial({ 1, 1, 1 });
//	vec3 final_colour({ 0.5, 0.7, 1.0 });
//	colour += attenuation * (t * initial + (1 - t) * final_colour);
//
//	return colour;
//
//}

__device__ bool castRay(const ray& r, hitRecord& record, objectList** objList) {
	float minDist = 1e6;
	float maxDist = 0.0;

	bool intersectionFound = false;
	hitRecord temp;
	for(int i = 0; i < (*objList)->numItems; i++) {
		//printf("HERE 4\n");

			//if(i == 1) {
			//	printf("PLANE\n");
			//	
			//}
		if((*objList)->list[i]->checkIntersection(r, temp)) {
			//if(i == 0) {
			//	//printf("HIT PLANE");
			//	record.localNormal = vec3({0, +1, 0});
			//	record.hitObj = world[i];
			//	record.localColour = vec3({1,1,1});
			//	return true;
			//}
			intersectionFound = true;
			float dist = length(temp.intPoint - r.m_point1);
			if(dist < minDist) {
				minDist = dist;
				record = temp;
				record.hitObj = (*objList)->list[i];
			}
		}
	}
	return intersectionFound;
}

__device__ vec3 pixel_colour(const int x, const int y,
	const int xSize, const int ySize,
	camera** cam,
	lightBase** d_lights, int lightSize,
	objectList** objList
) {
	//return vec3({0, 1, 0});

	float xFact = 1.0 / (static_cast<float>(xSize) / 2.0);
	float yFact = 1.0 / (static_cast<float>(ySize) / 2.0);


	float normX = (static_cast<float>(x) * xFact) - 1.0;
	float normY = (static_cast<float>(y) * yFact) - 1.0;

	ray cameraRay;

	(*cam)->generateRay(normX, normY, cameraRay);

	hitRecord record;

	// DEBUGGING


	// END DEBUGGING

	bool validInt = castRay(cameraRay, record, objList);

	if(validInt) {
		//return record.localColour;
		//printf("HIT PLANE \n");
		vec3 colour, difColour;
		float intensity = 0.0f;
		bool validIllum = d_lights[0]->computeIllumination(record, record.hitObj, (*objList)->list, (*objList)->numItems, colour, intensity);
		if(validIllum) {
			//colour *= intensity;
			//printf("%.2f\n", intensity);
			difColour = colour * intensity * record.localColour;
		}
		else {
			// Debugging shadows
			//return vec3({0, 1, 0});
		}
		return difColour;
	}
	else {
		//float vertSize = (*cam)->getVertSize();
		//auto z = cameraRay.m_AB.z;
		//float t = (z - ((*cam)->getScreenCentre().z - vertSize)) / (2 * vertSize);

		////float t = 0.5 * (-dir_unit_vec[2] + 1.0);
		////if(t < 0.0 or t > 1.0) {
		////	std::cerr << t << '\n';
		////	exit(0);
		////}

		//t = clamp(t, 0.0, 1.0);
		//vec3 initial({ 1.0, 1.0, 1.0 });
		//vec3 finl({ 0.5, 0.5, 1.0 });

		//vec3 res = t * initial + (1 - t) * finl;
		return vec3();
	}
}

__global__
void render(vec3* buff, const int xSize, const int ySize, camera** cam, lightBase** d_lights, int lightSize, objectList** objList) {
	//if(threadIdx.x == 0 && blockIdx.x == 0) {
	//	printf("HERE render<> %f\n", (*objList)->list[1]->m_baseColour.x);

	//}

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x >= xSize || y >= ySize) {
		return;
	}
	buff[y * xSize + x] = pixel_colour(x, y, xSize, ySize, cam, d_lights, lightSize, objList);
}

__global__
void create_world(objectList** objList, lightBase** d_lights, objectBase** d_world) {
	if(threadIdx.x == 0 && blockIdx.x == 0) {
		objectBase* sphere1 = new sphere();
		GTForm* sphereTform = new GTForm();
		sphereTform->setTransform(
			vec3({ 0.5, 0.5, 0.0 }),
			vec3({ 0.0, 0.0, 0.0 }),
			vec3({ 0.4, 0.4, 0.4 })
		);

		GTForm* planeTform = new GTForm();
		planeTform->setTransform(
			vec3({ 0.0, 0.0, -1.0 }),
			vec3({ 0.0, 0.0, 0.0 }),
			vec3({ 3.0, 3.0, 3.0 })
		);

		objectBase* plane1 = new plane();
		plane1->m_baseColour = vec3({ 0.5, 0.5, 0.5 });
		plane1->setTransformMatrix(planeTform);

		GTForm* sphereTform2 = new GTForm();
		sphereTform2->setTransform(
			vec3({ -2.5, 0.0, +0.2 }),
			vec3({ 0.0, 0.0, 0.0 }),
			vec3({ 0.5, 0.5, 0.5 })
		);
		sphere1->setTransformMatrix(sphereTform2);

		objectBase* sphere2 = new sphere();
		sphere2->m_baseColour = vec3({ 0.0, 0.7, 0.9 });
		sphere2->setTransformMatrix(sphereTform);

		(*objList) = new objectList(d_world, 5);

		//(*objList)->list[0] = sphere1;
		//(*objList)->list[1] = plane1;
		//(*objList)->list[2] = sphere2;
		objectBase* sphere3 = new sphere();
		sphere3->setTransformMatrix(planeTform);
		(*objList)->addItem(&sphere1);
		(*objList)->addItem(&plane1);
		(*objList)->addItem(&sphere2);

		//printf("HERE 2 %f\n", (*objList)->list[1]->m_baseColour.x);

		//world[0]->m_baseColour = vec3({ 0.0, 0.0, 1.0 });
		d_lights[0] = new pointLight();
		d_lights[0]->m_location = vec3({ 0.0, 0.0, +10.0 });
	}
}

__global__
void create_camera(camera** d_camera, vec3 cameraPosition, const int xSize, const int ySize) {
	if(threadIdx.x == 0 && blockIdx.x == 0) {
		//printf("CREATED CAMERA, xSize : %d , ySize : %d\n", xSize, ySize);
		if(!(*d_camera)) {
			(*d_camera) = new camera();
		}

		(*d_camera)->setPosition(cameraPosition);
		(*d_camera)->setLookAt(vec3{ 0.0, 0.0, 0.0 });
		(*d_camera)->setUp(vec3{ 0.0, 0.0, 1.0 });
		(*d_camera)->setHorzSize(0.25);
	}
}

__global__
void update_camera(camera** d_camera, const float dRight, const float dUp, const float dForward, const float theta, const float phi, const int xSize, const int ySize) {
	if(!(*d_camera)) {
		return;
	}
	(*d_camera)->setAspectRatio(((float) xSize) / ((float) ySize));
	(*d_camera)->UpdateCameraGeometry();
	(*d_camera)->moveRight(dRight);
	(*d_camera)->moveVertically(dUp);
	(*d_camera)->moveForward(dForward);
	(*d_camera)->rotateAlignment(theta, phi);
}



__global__
void free_world(camera** d_camera) {
	if(*d_camera) {
		delete* d_camera;
	}
}

namespace fRT {
	Scene::Scene() {
		m_changedState = true;
		m_worldChanged = true;
		m_cameraPosition = vec3({ 0, -15, 1 });
		m_hostImageBuffer = nullptr;


	}

	Scene::~Scene() {
		//checkCudaErrors1(hipFree(m_sharedImageBuffer));
		checkCudaErrors1(hipDeviceReset());
		delete[] m_hostImageBuffer;
		delete[] m_ImageData;
	}

	bool Scene::onInit() {
		// Allocate memory to GPU for entire application run time
		size_t numPixels = 1920 * 1080 * 2;
		size_t frame_buffer_size = numPixels * sizeof(vec3);

		checkCudaErrors1(hipMalloc((void**) &m_deviceImageBuffer, frame_buffer_size));

		// Create imageData at constructor
		m_ImageData = new uint32_t[numPixels];

		if(!m_hostImageBuffer) {
			m_hostImageBuffer = (vec3*) malloc(frame_buffer_size);
		}

		// Pre - allocate all memory for the pointers on GPU
		checkCudaErrors1(hipMalloc((void**) &d_camera, sizeof(camera*)));

		checkCudaErrors1(hipMalloc((void**) &d_objList, sizeof(objectList*)));
		checkCudaErrors1(hipMalloc((void**) &d_world, 6 * sizeof(objectBase*)));
		checkCudaErrors1(hipMalloc((void**) &d_lights, 6 * sizeof(lightBase*)));


		// Create the world on initialisation as the objects do not change
		// Might move later to change when updated
		create_world << <1, 1 >> > (d_objList, d_lights, d_world);
		checkCudaErrors1(hipGetLastError());
		checkCudaErrors1(hipDeviceSynchronize());

		// Create the camera here, update in Render if required
		create_camera << <1, 1 >> > (d_camera, m_cameraPosition, 1, 1);
		checkCudaErrors1(hipGetLastError());
		checkCudaErrors1(hipDeviceSynchronize());

		return true;
	}

	void Scene::OnResize(uint32_t width, uint32_t height) {
		if(m_FinalImage) {
			// No resize necessary
			if(m_FinalImage->GetWidth() == width && m_FinalImage->GetHeight() == height) {
				return;
			}

			m_FinalImage->Resize(width, height);
		}
		else {
			m_FinalImage = std::make_shared<Walnut::Image>(width, height, Walnut::ImageFormat::RGBA);
		}

		//delete[] m_ImageData;
		//m_ImageData = new uint32_t[width * height * 2];
	}

	// Function to actually render the image
	bool Scene::Render() {
		if(!doneInitialise) {
			onInit();
			doneInitialise = true;
		}

		//std::cerr << "RENDERING IMAGE" << std::endl;
		//auto start_time = std::chrono::high_resolution_clock::now();

		// Get dimensions of image
		int xSize = m_FinalImage->GetWidth();
		int ySize = m_FinalImage->GetHeight();

		//printf("xSize : %d, ySize : %d\n", xSize, ySize);

		// frame buffer size will control how many bytes or the size that will be copied back from
		// device memory as device memory size is constant
		size_t numPixels = xSize * 1ll * ySize;
		size_t frame_buffer_size = numPixels * sizeof(vec3);

		//checkCudaErrors1(hipMalloc((void**) &m_deviceImageBuffer, sizeof(vec3) * 1920 * 1080));
		int tx = 32;
		int ty = 32;


		if(m_resetCamera) {
			create_camera << <1, 1 >> > (d_camera, m_cameraPosition, 1, 1);
			checkCudaErrors1(hipGetLastError());
			checkCudaErrors1(hipDeviceSynchronize());
			m_resetCamera = false;
		}

		// Update the camera on device
		update_camera << <1, 1 >> > (d_camera, dRight, dUp, dForward, dTheta, dPhi, xSize, ySize);
		checkCudaErrors1(hipGetLastError());
		checkCudaErrors1(hipDeviceSynchronize());
		dUp = 0.0f;
		dRight = 0.0f;
		dForward = 0.0f;
		dTheta = 0.0f;
		dPhi = 0.0f;

		// Call create world kernel to make objects on the device
		int worldSize = 3;
		int lightSize = 1;

		// Render the buffer
		dim3 blocks(xSize / tx + 1, ySize / ty + 1);
		dim3 threads(tx, ty);
		render << <blocks, threads >> > (m_deviceImageBuffer, xSize, ySize, d_camera, d_lights, lightSize, d_objList);
		checkCudaErrors1(hipGetLastError());
		checkCudaErrors1(hipDeviceSynchronize());


		checkCudaErrors1(hipMemcpy(m_hostImageBuffer, m_deviceImageBuffer, frame_buffer_size, hipMemcpyDeviceToHost));
		//checkCudaErrors1(hipFree(m_deviceImageBuffer));

		//auto end_time = std::chrono::high_resolution_clock::now();
		//std::cerr << "\nTIME TAKEN: " << " ";
		//std::cerr << std::fixed << std::setprecision(3) << std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count() * 1e-3 << " s\n" << std::endl;
		for(int y = 0; y < ySize; y++) {
			for(int x = 0; x < xSize; x++) {
				size_t pixelIdx = y * xSize + x;
				//int pixelIdx = y * xSize * 3 + x * 3;
				float red = m_hostImageBuffer[pixelIdx].x;
				float green = m_hostImageBuffer[pixelIdx].y;
				float blue = m_hostImageBuffer[pixelIdx].z;

				unsigned char ir = static_cast<unsigned char>((red) * 255.0);
				unsigned char ig = static_cast<unsigned char>((green) * 255.0);
				unsigned char ib = static_cast<unsigned char>((blue) * 255.0);

				m_ImageData[pixelIdx] = (255 << 24) | (ib << 16) | (ig << 8) | (ir);
			}
		}

		m_FinalImage->SetData(m_ImageData);

		return true;
	}

	bool Scene::keyPressedW() {
		//m_cameraPosition.z += m_cameraStep;
		dForward += m_cameraStep;
		m_changedState = true;
		return true;
	}

	bool Scene::keyPressedA() {
		//m_cameraPosition.x -= m_cameraStep;
		dRight -= m_cameraStep;
		m_changedState = true;
		return true;
	}

	bool Scene::keyPressedS() {
		//m_cameraPosition.z -= m_cameraStep;
		dForward -= m_cameraStep;
		m_changedState = true;
		return true;
	}

	bool Scene::keyPressedD() {
		//m_cameraPosition.x += m_cameraStep;
		dRight += m_cameraStep;
		m_changedState = true;
		return true;
	}

	bool Scene::keyPressedUpArrow() {
		dPhi += m_cameraAngleStep;
		m_changedState = true;
		return true;
	}

	bool Scene::keyPressedDownArrow() {
		dPhi -= m_cameraAngleStep;
		m_changedState = true;
		return true;
	}

	bool Scene::keyPressedLeftArrow() {
		dTheta -= m_cameraAngleStep;
		m_changedState = true;
		return true;
	}

	bool Scene::keyPressedRightArrow() {
		dTheta += m_cameraAngleStep;
		m_changedState = true;
		return true;
	}

	bool Scene::keyPressedR() {
		m_resetCamera = true;
		return true;
	}

	bool Scene::keyPressedSpace() {
		dUp += m_cameraStep;
		m_changedState = true;
		return true;
	}

	bool Scene::keyPressedLCtrl() {
		dUp -= m_cameraStep;
		m_changedState = true;
		return true;
	}

};

